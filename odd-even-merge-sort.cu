#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<conio.h>
#include <random>
#include <stdint.h>
#include <hip/driver_types.h >

static __device__ __inline__ uint32_t __mysmid(){
	uint32_t smid;
	asm volatile("mov.u32 %0 , %%smid;" : "=r"(smid));
	return smid;
}

static __device__ __inline__ uint32_t __mywarpid(){
	uint32_t warpid;
	asm volatile("mov.u32 %0 , %%warpid;" : "=r"(warpid));
	return warpid;
}

static __device__ __inline__ uint32_t __mylaneid(){
	uint32_t laneid;
	asm volatile("mov.u32 %0 , %%laneid;" : "=r"(laneid));
	return laneid;
}


__global__ void odd(int *arr,int n){
  	int i=threadIdx.x;
  	int temp;
  	if(i%2==1&&i<n-1){
  	if(arr[i]>arr[i+1])
  	{
  		temp=arr[i];
  		arr[i]=arr[i+1];
  		arr[i+1]=temp;
  	}
  	printf("Odd thread %d SMID=%d warp ID=%d warp lane ID=%d \n",i,__mysmid(),__mywarpid(),__mylaneid());
  	}
}

__global__ void even(int *arr,int n){
  	int i=threadIdx.x;
  	int temp;
  	if(i%2==0&&i<n-1){
  	if(arr[i]>arr[i+1])
  	{
  		temp=arr[i];
  		arr[i]=arr[i+1];
  		arr[i+1]=temp;
  	}
  	printf("Even thread %d SMID=%d warp ID=%d warp lane ID=%d \n",i,__mysmid(),__mywarpid(),__mylaneid());
  	}
}

int main(){
	int SIZE,k,*A,p,j;
	int *d_A;
	float time;
	hipEvent_t start, stop;
	
	printf("Enter the size of the array\n");
	scanf("%d",&SIZE);
	A=(int *)malloc(SIZE*sizeof(int));
	hipMalloc(&d_A,SIZE*sizeof(int));
	for(k=0;k<SIZE;k++)
		//scanf("%d",&A[k]);
		A[k]=rand()%1000;
		
	
	hipMemcpy(d_A,A,SIZE*sizeof(int),hipMemcpyHostToDevice);
	if(SIZE%2==0)
		p=SIZE/2;
	else
		p=SIZE/2+1;

	hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

	for(j=0;j<p;j++){
		even<<<3,SIZE>>>(d_A,SIZE);
		if(j!=p-1)
			odd<<<3,SIZE>>>(d_A,SIZE);
		if(j==p-1&&SIZE%2==0)
			odd<<<1,SIZE>>>(d_A,SIZE);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipMemcpy(A,d_A,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	for(k=0;k<SIZE;k++)
		printf("%d ",A[k]);
	
	printf("\nTime to generate:  %3.1f ms \n", time);
	free(A);
	hipFree(d_A);
	
	getch();
	
}
